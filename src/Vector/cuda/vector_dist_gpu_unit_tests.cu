#include "hip/hip_runtime.h"

#define BOOST_TEST_DYN_LINK
#include <boost/test/unit_test.hpp>
#include "VCluster/VCluster.hpp"
#include <Vector/vector_dist.hpp>
#include "Vector/tests/vector_dist_util_unit_tests.hpp"

#define SUB_UNIT_FACTOR 1024

BOOST_AUTO_TEST_SUITE( vector_dist_gpu_test )

void print_test(std::string test, size_t sz)
{
	if (create_vcluster().getProcessUnitID() == 0)
		std::cout << test << " " << sz << "\n";
}


__global__  void initialize_props(vector_dist_ker<3, float, aggregate<float, float [3], float[3]>> vd)
{
	auto p = GET_PARTICLE(vd);

	vd.template getProp<0>(p) = vd.getPos(p)[0] + vd.getPos(p)[1] + vd.getPos(p)[2];

	vd.template getProp<1>(p)[0] = vd.getPos(p)[0] + vd.getPos(p)[1];
	vd.template getProp<1>(p)[1] = vd.getPos(p)[0] + vd.getPos(p)[2];
	vd.template getProp<1>(p)[2] = vd.getPos(p)[1] + vd.getPos(p)[2];
}

template<typename CellList_type>
__global__  void calculate_force(vector_dist_ker<3, float, aggregate<float, float[3], float [3]>> vd,
		                         vector_dist_ker<3, float, aggregate<float, float[3], float [3]>> vd_sort,
		                         CellList_type cl)
{
	auto p = GET_PARTICLE(vd);

	Point<3,float> xp = vd.getPos(p);

    auto it = cl.getNNIterator(cl.getCell(xp));

    auto cell = cl.getCell(xp);

    Point<3,float> force1({0.0,0.0,0.0});
    Point<3,float> force2({0.0,0.0,0.0});

    while (it.isNext())
    {
    	auto q1 = it.get();
    	auto q2 = it.get_orig();

    	if (q2 == p) {++it; continue;}

    	Point<3,float> xq_1 = vd_sort.getPos(q1);
    	Point<3,float> xq_2 = vd.getPos(q2);

    	Point<3,float> r1 = xq_1 - xp;
    	Point<3,float> r2 = xq_2 - xp;

    	// Normalize

    	r1 /= r1.norm();
    	r2 /= r2.norm();

    	force1 += vd_sort.template getProp<0>(q1)*r1;
    	force2 += vd.template getProp<0>(q2)*r2;

    	++it;
    }

    vd.template getProp<1>(p)[0] = force1.get(0);
    vd.template getProp<1>(p)[1] = force1.get(1);
    vd.template getProp<1>(p)[2] = force1.get(2);

    vd.template getProp<2>(p)[0] = force2.get(0);
    vd.template getProp<2>(p)[1] = force2.get(1);
    vd.template getProp<2>(p)[2] = force2.get(2);
}

template<typename CellList_type>
__global__  void calculate_force_full_sort(vector_dist_ker<3, float, aggregate<float, float[3], float [3]>> vd,
		                         	 	   CellList_type cl)
{
	auto p = GET_PARTICLE(vd);

	Point<3,float> xp = vd.getPos(p);

    auto it = cl.getNNIterator(cl.getCell(xp));

    auto cell = cl.getCell(xp);

    Point<3,float> force1({0.0,0.0,0.0});

    while (it.isNext())
    {
    	auto q1 = it.get();

    	if (q1 == p) {++it; continue;}

    	Point<3,float> xq_1 = vd.getPos(q1);

    	Point<3,float> r1 = xq_1 - xp;

    	// Normalize

    	r1 /= r1.norm();

    	force1 += vd.template getProp<0>(q1)*r1;

    	++it;
    }

    vd.template getProp<1>(p)[0] = force1.get(0);
    vd.template getProp<1>(p)[1] = force1.get(1);
    vd.template getProp<1>(p)[2] = force1.get(2);
}

template<typename CellList_type, typename vector_type>
bool check_force(CellList_type & NN_cpu, vector_type & vd)
{
	auto it6 = vd.getDomainIterator();

	bool match = true;

	while (it6.isNext())
	{
		auto p = it6.get();

		Point<3,float> xp = vd.getPos(p);

		// Calculate on CPU

		Point<3,float> force({0.0,0.0,0.0});

		auto NNc = NN_cpu.getNNIterator(NN_cpu.getCell(xp));

		while (NNc.isNext())
		{
			auto q = NNc.get();

	    	if (q == p.getKey()) {++NNc; continue;}

	    	Point<3,float> xq_2 = vd.getPos(q);
	    	Point<3,float> r2 = xq_2 - xp;

	    	// Normalize

	    	r2 /= r2.norm();
	    	force += vd.template getProp<0>(q)*r2;

			++NNc;
		}

		match &= fabs(vd.template getProp<1>(p)[0] - vd.template getProp<2>(p)[0]) < 0.0001;
		match &= fabs(vd.template getProp<1>(p)[1] - vd.template getProp<2>(p)[1]) < 0.0001;
		match &= fabs(vd.template getProp<1>(p)[2] - vd.template getProp<2>(p)[2]) < 0.0001;

		match &= fabs(vd.template getProp<1>(p)[0] - force.get(0)) < 0.0001;
		match &= fabs(vd.template getProp<1>(p)[1] - force.get(1)) < 0.0001;
		match &= fabs(vd.template getProp<1>(p)[2] - force.get(2)) < 0.0001;

		++it6;
	}

	return match;
}

BOOST_AUTO_TEST_CASE( vector_dist_gpu_ghost_get )
{
	auto & v_cl = create_vcluster();

	if (v_cl.size() > 16)
	{return;}

	Box<3,float> domain({0.0,0.0,0.0},{1.0,1.0,1.0});

	// set the ghost based on the radius cut off (make just a little bit smaller than the spacing)
	Ghost<3,float> g(0.1);

	// Boundary conditions
	size_t bc[3]={PERIODIC,PERIODIC,PERIODIC};

	vector_dist_gpu<3,float,aggregate<float,float[3],float[3]>> vd(1000,domain,bc,g);

	auto it = vd.getDomainIterator();

	while (it.isNext())
	{
		auto p = it.get();

		vd.getPos(p)[0] = (float)rand() / RAND_MAX;
		vd.getPos(p)[1] = (float)rand() / RAND_MAX;
		vd.getPos(p)[2] = (float)rand() / RAND_MAX;

		vd.template getProp<0>(p) = vd.getPos(p)[0] + vd.getPos(p)[1] + vd.getPos(p)[2];

		vd.template getProp<1>(p)[0] = vd.getPos(p)[0] + vd.getPos(p)[1];
		vd.template getProp<1>(p)[1] = vd.getPos(p)[0] + vd.getPos(p)[2];
		vd.template getProp<1>(p)[2] = vd.getPos(p)[1] + vd.getPos(p)[2];

		vd.template getProp<2>(p)[0] = vd.getPos(p)[0] + 3.0*vd.getPos(p)[1];
		vd.template getProp<2>(p)[1] = vd.getPos(p)[0] + 3.0*vd.getPos(p)[2];
		vd.template getProp<2>(p)[2] = vd.getPos(p)[1] + 3.0*vd.getPos(p)[2];


		++it;
	}

	// Ok we redistribute the particles (CPU based)
	vd.map();

	vd.template ghost_get<0,1,2>();

	// Now we check the the ghost contain the correct information

	bool check = true;

	auto itg = vd.getDomainAndGhostIterator();

	while (itg.isNext())
	{
		auto p = itg.get();

		check &= (vd.template getProp<0>(p) == vd.getPos(p)[0] + vd.getPos(p)[1] + vd.getPos(p)[2]);

		check &= (vd.template getProp<1>(p)[0] == vd.getPos(p)[0] + vd.getPos(p)[1]);
		check &= (vd.template getProp<1>(p)[1] == vd.getPos(p)[0] + vd.getPos(p)[2]);
		check &= (vd.template getProp<1>(p)[2] == vd.getPos(p)[1] + vd.getPos(p)[2]);

		check &= (vd.template getProp<2>(p)[0] == vd.getPos(p)[0] + 3.0*vd.getPos(p)[1]);
		check &= (vd.template getProp<2>(p)[1] == vd.getPos(p)[0] + 3.0*vd.getPos(p)[2]);
		check &= (vd.template getProp<2>(p)[2] == vd.getPos(p)[1] + 3.0*vd.getPos(p)[2]);

		++itg;
	}

	size_t tot_s = vd.size_local_with_ghost();

	v_cl.sum(tot_s);
	v_cl.execute();

	// We check that we check something
	BOOST_REQUIRE(tot_s > 1000);
}

BOOST_AUTO_TEST_CASE( vector_dist_gpu_test)
{
	auto & v_cl = create_vcluster();

	if (v_cl.size() > 16)
	{return;}

	Box<3,float> domain({0.0,0.0,0.0},{1.0,1.0,1.0});

	// set the ghost based on the radius cut off (make just a little bit smaller than the spacing)
	Ghost<3,float> g(0.1);

	// Boundary conditions
	size_t bc[3]={NON_PERIODIC,NON_PERIODIC,NON_PERIODIC};

	vector_dist_gpu<3,float,aggregate<float,float[3],float[3]>> vd(1000,domain,bc,g);

	auto it = vd.getDomainIterator();

	while (it.isNext())
	{
		auto p = it.get();

		vd.getPos(p)[0] = (float)rand() / RAND_MAX;
		vd.getPos(p)[1] = (float)rand() / RAND_MAX;
		vd.getPos(p)[2] = (float)rand() / RAND_MAX;

		++it;
	}

	// Ok we redistribute the particles (CPU based)
	vd.map();

	size_t size_l = vd.size_local();

	v_cl.sum(size_l);
	v_cl.execute();

	BOOST_REQUIRE_EQUAL(size_l,1000);


	auto & ct = vd.getDecomposition();

	bool noOut = true;
	size_t cnt = 0;

	auto it2 = vd.getDomainIterator();

	while (it2.isNext())
	{
		auto p = it2.get();

		noOut &= ct.isLocal(vd.getPos(p));

		cnt++;
		++it2;
	}

	BOOST_REQUIRE_EQUAL(noOut,true);
	BOOST_REQUIRE_EQUAL(cnt,vd.size_local());

	vd.write("test_out_gpu");

	// now we offload all the properties

	auto it3 = vd.getDomainIteratorGPU();

	// offload to device
	vd.hostToDevicePos();

	initialize_props<<<it3.wthr,it3.thr>>>(vd.toKernel());

	// now we check what we initialized

	vd.deviceToHostProp<0,1>();

	auto it4 = vd.getDomainIterator();

	while (it4.isNext())
	{
		auto p = it4.get();

		BOOST_REQUIRE_CLOSE(vd.template getProp<0>(p),vd.getPos(p)[0] + vd.getPos(p)[1] + vd.getPos(p)[2],0.01);

		BOOST_REQUIRE_CLOSE(vd.template getProp<1>(p)[0],vd.getPos(p)[0] + vd.getPos(p)[1],0.01);
		BOOST_REQUIRE_CLOSE(vd.template getProp<1>(p)[1],vd.getPos(p)[0] + vd.getPos(p)[2],0.01);
		BOOST_REQUIRE_CLOSE(vd.template getProp<1>(p)[2],vd.getPos(p)[1] + vd.getPos(p)[2],0.01);

		//std::cout << "PROP 0 " << vd.template getProp<0>(p) << "   " << vd.getPos(p)[0] + vd.getPos(p)[1] + vd.getPos(p)[2] << std::endl;

		++it4;
	}

	// here we do a ghost_get
	vd.ghost_get<0>();

	// we re-offload what we received
	vd.hostToDevicePos();
	vd.template hostToDeviceProp<0>();

	auto NN = vd.getCellListGPU(0.1);
	auto NN_cpu = vd.getCellList(0.1);

	auto it5 = vd.getDomainIteratorGPU();

	calculate_force<decltype(NN.toKernel())><<<it5.wthr,it5.thr>>>(vd.toKernel(),vd.toKernel_sorted(),NN.toKernel());

	vd.template deviceToHostProp<1,2>();

	bool test = check_force(NN_cpu,vd);
	BOOST_REQUIRE_EQUAL(test,true);

	// We do exactly the same test as before, but now we completely use the sorted version

	calculate_force_full_sort<decltype(NN.toKernel())><<<it5.wthr,it5.thr>>>(vd.toKernel_sorted(),NN.toKernel());

	vd.template deviceToHostProp<1>();

	test = check_force(NN_cpu,vd);
	BOOST_REQUIRE_EQUAL(test,true);

	// check

	// Now we do a ghost_get from CPU

	// Than we offload on GPU

	// We construct a Cell-list

	// We calculate force on CPU and GPU to check if they match



}

BOOST_AUTO_TEST_CASE( vector_dist_map_on_gpu_test)
{
	auto & v_cl = create_vcluster();

	if (v_cl.size() > 16)
	{return;}

	Box<3,float> domain({0.0,0.0,0.0},{1.0,1.0,1.0});

	// set the ghost based on the radius cut off (make just a little bit smaller than the spacing)
	Ghost<3,float> g(0.1);

	// Boundary conditions
	size_t bc[3]={NON_PERIODIC,NON_PERIODIC,NON_PERIODIC};

	vector_dist_gpu<3,float,aggregate<float,float[3],float[3]>> vd(1000,domain,bc,g);

	auto it = vd.getDomainIterator();

	while (it.isNext())
	{
		auto p = it.get();

		vd.getPos(p)[0] = (float)rand() / RAND_MAX;
		vd.getPos(p)[1] = (float)rand() / RAND_MAX;
		vd.getPos(p)[2] = (float)rand() / RAND_MAX;

		vd.template getProp<0>(p) = vd.getPos(p)[0] + vd.getPos(p)[1] + vd.getPos(p)[2];

		vd.template getProp<1>(p)[0] = vd.getPos(p)[0];
		vd.template getProp<1>(p)[1] = vd.getPos(p)[1];
		vd.template getProp<1>(p)[2] = vd.getPos(p)[2];

		vd.template getProp<2>(p)[0] = vd.getPos(p)[0] + vd.getPos(p)[1];
		vd.template getProp<2>(p)[1] = vd.getPos(p)[0] + vd.getPos(p)[2];
		vd.template getProp<2>(p)[2] = vd.getPos(p)[1] + vd.getPos(p)[2];

		++it;
	}

	// move on device
	vd.hostToDevicePos();
	vd.hostToDeviceProp<0,1,2>();

	// Ok we redistribute the particles (GPU based)
	vd.map(RUN_ON_DEVICE);

	// Reset the host part

	auto it3 = vd.getDomainIterator();

	while (it3.isNext())
	{
		auto p = it3.get();

		vd.getPos(p)[0] = 1.0;
		vd.getPos(p)[1] = 1.0;
		vd.getPos(p)[2] = 1.0;

		vd.template getProp<0>(p) = 0.0;

		vd.template getProp<0>(p) = 0.0;
		vd.template getProp<0>(p) = 0.0;
		vd.template getProp<0>(p) = 0.0;

		vd.template getProp<0>(p) = 0.0;
		vd.template getProp<0>(p) = 0.0;
		vd.template getProp<0>(p) = 0.0;

		++it3;
	}

	// we move from Device to CPU

	vd.deviceToHostPos();
	vd.deviceToHostProp<0,1,2>();

	// Check

	auto it2 = vd.getDomainIterator();

	bool match = true;
	while (it2.isNext())
	{
		auto p = it2.get();

		match &= vd.template getProp<0>(p) == vd.getPos(p)[0] + vd.getPos(p)[1] + vd.getPos(p)[2];

		match &= vd.template getProp<1>(p)[0] == vd.getPos(p)[0];
		match &= vd.template getProp<1>(p)[1] == vd.getPos(p)[1];
		match &= vd.template getProp<1>(p)[2] == vd.getPos(p)[2];

		match &= vd.template getProp<2>(p)[0] == vd.getPos(p)[0] + vd.getPos(p)[1];
		match &= vd.template getProp<2>(p)[1] == vd.getPos(p)[0] + vd.getPos(p)[2];
		match &= vd.template getProp<2>(p)[2] == vd.getPos(p)[1] + vd.getPos(p)[2];

		++it2;
	}

	BOOST_REQUIRE_EQUAL(match,true);

	// count local particles

	size_t l_cnt = 0;
	size_t nl_cnt = 0;
	size_t n_out = 0;

	// Domain + ghost box
	Box<3,float> dom_ext = domain;
	dom_ext.enlarge(g);

	auto it5 = vd.getDomainIterator();
	count_local_n_local<3>(vd,it5,bc,domain,dom_ext,l_cnt,nl_cnt,n_out);

	BOOST_REQUIRE_EQUAL(n_out,0);
	BOOST_REQUIRE_EQUAL(l_cnt,vd.size_local());

	vd.write("gpu_write_test");

	vd.ghost_get<0,1,2>(RUN_ON_DEVICE);
}

BOOST_AUTO_TEST_SUITE_END()
