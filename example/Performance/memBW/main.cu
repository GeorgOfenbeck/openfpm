
#include <hip/hip_runtime.h>
#ifdef __NVCC__

#include "Vector/map_vector.hpp"
#include "util/stat/common_statistics.hpp"

//! Memory bandwidth with small calculations
template<typename vector_type, typename vector_type2>
__global__ void translate_fill_prop_write(vector_type vd_out, vector_type2 vd_in)
{
	auto p = blockIdx.x * blockDim.x + threadIdx.x;

	float a = vd_in.template get<0>(p)[0];

	vd_out.template get<0>(p) = a;

	vd_out.template get<1>(p)[0] = a;
	vd_out.template get<1>(p)[1] = a;

	vd_out.template get<2>(p)[0][0] = a;
	vd_out.template get<2>(p)[0][1] = a;
	vd_out.template get<2>(p)[1][0] = a;
    vd_out.template get<2>(p)[1][1] = a;
    vd_in.template get<0>(p)[1] = a;
}


template<typename vector_type, typename vector_type2>
__global__ void translate_fill_prop_read(vector_type vd_out, vector_type2 vd_in)
{
	auto p = blockIdx.x * blockDim.x + threadIdx.x;

	float a = vd_out.template get<0>(p);

	float b = vd_out.template get<1>(p)[0];
	float c = vd_out.template get<1>(p)[1];

	float d = vd_out.template get<2>(p)[0][0];
	float e = vd_out.template get<2>(p)[0][1];
	float f = vd_out.template get<2>(p)[1][0];
	float g = vd_out.template get<2>(p)[1][1];
    
	float h = vd_in.template get<0>(p)[0];
	vd_in.template get<0>(p)[1] = a+b+c+d+e+f+g+h;
}

// Arrays

__global__ void translate_fill_prop_write_array(float * vd_out_scal,
                                                       float * vd_out_vec,
                                                       float * vd_out_mat,
                                                       float * vd_in_vec,
                                                       int stride)
{
	auto p = blockIdx.x * blockDim.x + threadIdx.x;

	float a = vd_in_vec[p* + 0*stride];
	float b = vd_in_vec[p* + 1*stride];

	vd_out_scal[p] = a + b;

	vd_out_vec[p + 0*stride] = a;
	vd_out_vec[p + 1*stride] = b;

	vd_out_mat[p + 0*2*stride + 0*stride ] = a;
	vd_out_mat[p + 0*2*stride + 1*stride ] = b;
	vd_out_mat[p + 1*2*stride + 0*stride ] = a + b;
	vd_out_mat[p + 1*2*stride + 1*stride ] = b - a;
}


template<typename vector_type, typename vector_type2>
__global__ void translate_fill_prop_read_array(vector_type vd_out, vector_type2 vd_in)
{
	auto p = blockIdx.x * blockDim.x + threadIdx.x;

	float a = vd_out.template get<0>(p);

	float b = vd_out.template get<1>(p)[0];
	float c = vd_out.template get<1>(p)[1];

	float d = vd_out.template get<2>(p)[0][0];
	float e = vd_out.template get<2>(p)[0][1];
	float f = vd_out.template get<2>(p)[1][0];
	float g = vd_out.template get<2>(p)[1][1];
    
	float h = vd_in.template get<0>(p)[0];
	vd_in.template get<0>(p)[1] = a+b+c+d+e+f+g+h;
}

template<typename in_type, typename out_type>
void check_write(in_type & in, out_type & out)
{
    out.template deviceToHost<0,1,2>();
    in.template deviceToHost<0>();

    bool success = true;
    for (int i = 0 ; i < 16777216 ; i++)
    {
        float a = in.template get<0>(i)[0];

        success &= out.template get<0>(i) == a;

        success &= out.template get<1>(i)[0] == a;
        success &= out.template get<1>(i)[1] == a;

        success &= out.template get<2>(i)[0][0] == a;
        success &= out.template get<2>(i)[0][1] == a;
        success &= out.template get<2>(i)[1][0] == a;
        success &= out.template get<2>(i)[1][1] == a;

        success &= in.template get<0>(i)[1] == a;
    }

    if (success == false)
    {
            std::cout << "FAIL WRITE" << std::endl;
            exit(1);
    }
}

template<typename in_type, typename out_type>
void check_read(in_type & in, out_type & out)
{
    out.template deviceToHost<0,1,2>();
    in.template deviceToHost<0>();

    bool success = true;
    for (int i = 0 ; i < 16777216 ; i++)
    {
        float a = out.template get<0>(i);

        float b = out.template get<1>(i)[0];
        float c = out.template get<1>(i)[1];

        float d = out.template get<2>(i)[0][0];
        float e = out.template get<2>(i)[0][1];
        float f = out.template get<2>(i)[1][0];
        float g = out.template get<2>(i)[1][1];

        float h = in.template get<0>(i)[0];

        success &= in.template get<0>(i)[1] == (a+b+c+d+e+f+g+h);

        if (success == false)
        {
            std::cout << "FAIL READ " << i << "   " << in.template get<0>(i)[1] << " != " << a+b+c+d+e+f+g+h << std::endl;
            exit(1);
        }
    }
}

template<typename vin_type, typename vout_type>
void initialize_buf(vin_type in, vout_type out)
{
    for (int i = 0 ; i < 16777216 ; i++)
    {
        in.template get<0>(i)[0] = i;
        in.template get<0>(i)[1] = i+100.0;

        out.template get<0>(i) = i+200.0;

        out.template get<1>(i)[0] = i;
        out.template get<1>(i)[1] = i+100.0;

        out.template get<2>(i)[0][0] = i;
        out.template get<2>(i)[0][1] = i+100.0;
        out.template get<2>(i)[1][0] = i+200.0;
        out.template get<2>(i)[1][1] = i+300.0;
    }

}

int main(int argc, char *argv[])
{
    init_wrappers();

    openfpm::vector_gpu<aggregate<float,float[2],float[2][2]>> out;
    openfpm::vector_gpu<aggregate<float[2]>> in;

    int nele = 16777216;

    out.resize(nele);
    in.resize(nele);

    initialize_buf(in,out);

    // Read write test with TLS

    auto ite = out.getGPUIterator(256);

    openfpm::vector<double> res;
    res.resize(100);

    in.hostToDevice<0>();

    for (int i = 0 ; i < 110 ; i++)
    {
        cudaDeviceSynchronize();
        timer t;
        t.start();


        CUDA_LAUNCH(translate_fill_prop_write,ite,out.toKernel(),in.toKernel());

        cudaDeviceSynchronize();

        t.stop();

        if (i >=10)
        {res.get(i-10) = nele*4*9 / t.getwct() * 1e-9;}

        std::cout << "Time: " << t.getwct() << std::endl;
        std::cout << "BW: " << nele*4*9 / t.getwct() * 1e-9 << " GB/s"  << std::endl;
    }

    double mean_write_tls = 0.0;
    double dev_write_tls = 0.0;
    standard_deviation(res,mean_write_tls,dev_write_tls);

    check_write(in,out);

    initialize_buf(in,out);

    for (int i = 0 ; i < 110 ; i++)
    {
        cudaDeviceSynchronize();
        timer t;
        t.start();


        CUDA_LAUNCH(translate_fill_prop_read,ite,out.toKernel(),in.toKernel());

        cudaDeviceSynchronize();

        t.stop();

        if (i >=10)
        {res.get(i-10) = nele*4*9 / t.getwct() * 1e-9;}

        std::cout << "Time: " << t.getwct() << std::endl;
        std::cout << "BW: " << nele*4*9 / t.getwct() * 1e-9 << " GB/s"  << std::endl;
    }

    double mean_read_tls = 0.0;
    double dev_read_tls = 0.0;
    standard_deviation(res,mean_read_tls,dev_read_tls);

    check_read(in,out);

    //////////////

    /////////////////////////////////////////// LAMBDA //////////////////////////////////////////

    initialize_buf(in,out);

    for (int i = 0 ; i < 110 ; i++)
    {
        cudaDeviceSynchronize();
        timer t;
        t.start();

        auto vd_out = out.toKernel();
        auto vd_in = in.toKernel();

        auto lamb = [vd_out,vd_in] __device__ (dim3 & blockIdx, dim3 & threadIdx)
        {
            auto p = blockIdx.x * blockDim.x + threadIdx.x;

            float a = vd_in.template get<0>(p)[0];
            float b = vd_in.template get<0>(p)[1];

	    vd_out.template get<0>(p) = a + b;

            vd_out.template get<1>(p)[0] = a;
            vd_out.template get<1>(p)[1] = b;

            vd_out.template get<2>(p)[0][0] = a;
            vd_out.template get<2>(p)[0][1] = b;
            vd_out.template get<2>(p)[1][0] = a + b;
            vd_out.template get<2>(p)[1][1] = b - a;
        };

        CUDA_LAUNCH_LAMBDA(ite, lamb);

        cudaDeviceSynchronize();

        t.stop();

        if (i >=10)
        {res.get(i-10) = nele*4*9 / t.getwct() * 1e-9;}

        std::cout << "Time: " << t.getwct() << std::endl;
        std::cout << "BW: " << nele*4*9 / t.getwct() * 1e-9 << " GB/s"  << std::endl;
    }

    double mean_write_lamb = 0.0;
    double dev_write_lamb = 0.0;
    standard_deviation(res,mean_write_lamb,dev_write_lamb);

    initialize_buf(in,out);

    for (int i = 0 ; i < 110 ; i++)
    {
        cudaDeviceSynchronize();
        timer t;
        t.start();


        auto vd_out = out.toKernel();
        auto vd_in = in.toKernel();

        auto lamb = [vd_out,vd_in] __device__ (dim3 & blockIdx, dim3 & threadIdx)
                            {
                                auto p = blockIdx.x * blockDim.x + threadIdx.x;

                                float a = vd_out.template get<0>(p);
                            
                                float b = vd_out.template get<1>(p)[0];
                                float c = vd_out.template get<1>(p)[1];
                            
                                float d = vd_out.template get<2>(p)[0][0];
                                float e = vd_out.template get<2>(p)[0][1];
                                float f = vd_out.template get<2>(p)[1][0];
                                float g = vd_out.template get<2>(p)[1][1];
                                
                                vd_in.template get<0>(p)[0] = a+b+c+d;
                                vd_in.template get<0>(p)[1] = e+f+g;
                            };

        CUDA_LAUNCH_LAMBDA(ite, lamb);

        cudaDeviceSynchronize();

        t.stop();

        if (i >=10)
        {res.get(i-10) = nele*4*9 / t.getwct() * 1e-9;}

        std::cout << "Time: " << t.getwct() << std::endl;
        std::cout << "BW: " << nele*4*9 / t.getwct() * 1e-9 << " GB/s"  << std::endl;
    }

    double mean_read_lamb = 0.0;
    double dev_read_lamb = 0.0;
    standard_deviation(res,mean_read_lamb,dev_read_lamb);

    #ifdef CUDIFY_USE_CUDA

    for (int i = 0 ; i < 110 ; i++)
    {
        cudaDeviceSynchronize();
        timer t;
        t.start();

        float * a = (float *)in.getDeviceBuffer<0>();
        float * b = (float *)out.getDeviceBuffer<1>();

        cudaMemcpy(a,b,2*16777216*4,cudaMemcpyDeviceToDevice);

        cudaDeviceSynchronize();

        t.stop();

        if (i >=10)
        {res.get(i-10) = nele*4*4 / t.getwct() * 1e-9;}

        std::cout << "Time: " << t.getwct() << std::endl;
        std::cout << "BW: " << nele*4*4 / t.getwct() * 1e-9 << " GB/s"  << std::endl;
    }    

    double mean_read_mes = 0.0;
    double dev_read_mes = 0.0;
    standard_deviation(res,mean_read_mes,dev_read_mes);

    std::cout << "Average measured: " << mean_read_mes << "  deviation: " << dev_read_mes << std::endl;

    #endif

    std::cout << "Average READ with TLS: " << mean_read_tls << "  deviation: " << dev_read_tls << std::endl;
    std::cout << "Average WRITE with TLS: " << mean_write_tls << "  deviation: " << dev_write_tls << std::endl;

    std::cout << "Average READ with lamb: " << mean_read_lamb << "  deviation: " << dev_read_lamb << std::endl;
    std::cout << "Average WRITE with lamb: " << mean_write_lamb << "  deviation: " << dev_write_lamb << std::endl;

}

#else

int main(int argc, char *argv[])
{
}

#endif

