//#define VCLUSTER_PERF_REPORT <------ Activate telemetry for the VCluster data-structure
//#define SYNC_BEFORE_TAKE_TIME <------ Force synchronization of the kernels everytime we take the time with the structure timer.
//                                      Use this option for telemetry and GPU otherwise the result are unreliable                                        
//#define ENABLE_GRID_DIST_ID_PERF_STATS  <------ Activate telementry for the grid data-structure

#include "Decomposition/Distribution/BoxDistribution.hpp"
#include "Grid/grid_dist_id.hpp"
#include "data_type/aggregate.hpp"
#include "timer.hpp"

/*!
 *
 * \page Grid_3_gs_3D_sparse_gpu_opt Gray Scott in 3D using sparse grids on GPU (Optimized)
 *
 * [TOC]
 *
 * # Solving a gray scott-system in 3D using Sparse grids on gpu (Optimized) # {#e3_gs_gray_scott_gpu}
 *
 * This example show how to solve a Gray-Scott system in 3D using sparse grids on gpu
 *
 * In figure is the final solution of the problem
 *
 * \htmlonly
 * <img src="http://ppmcore.mpi-cbg.de/web/images/examples/gray_scott_3d/gs_alpha.png"/>
 * \endhtmlonly
 *
 * More or less this example is the adaptation of the dense example in 3D
 *
 * \see \ref Grid_3_gs_3D
 *
 * # Optimizations
 *
 * Instead of using the default decomposition algorithm based on parmetis we use BoxDistribution. This decomposition divide the space equally 
 * across processors. The way to use a different algorithm for decomposing the sparse grid is given by changing the type of the Sparse grid
 * 
 * \snippet SparseGrid/2_gray_scott_3d_sparse_gpu_opt/main.cu grid definition
 *
 * Because the geometry is fixed we are also using the option SKIP_LABELLING. With this option active after a normal ghost_get we are able to 
 * activate certain optimization patterns in constructions of the sending buffers and merging data.
 *
 */

#ifdef __NVCC__

constexpr int U = 0;
constexpr int V = 1;

constexpr int U_next = 2;
constexpr int V_next = 3;

constexpr int x = 0;
constexpr int y = 1;
constexpr int z = 2;

//! \cond [grid definition] \endcond

typedef CartDecomposition<3,float, CudaMemory, memory_traits_inte, BoxDistribution<3,float> > Dec;

typedef sgrid_dist_id_gpu<3,float,aggregate<float,float,float,float>,CudaMemory, Dec> SparseGridType;

//! \cond [grid definition] \endcond

void init(SparseGridType & grid, Box<3,float> & domain)
{
	//! \cond [create points] \endcond

	typedef typename GetAddBlockType<SparseGridType>::type InsertBlockT;

	grid.addPoints([] __device__ (int i, int j, int k)
			        {
						return true;
			        },
			        [] __device__ (InsertBlockT & data, int i, int j, int k)
			        {
			        	data.template get<U>() = 1.0;
			        	data.template get<V>() = 0.0;
			        }
			        );


	grid.template flush<smax_<U>,smax_<V>>(flush_type::FLUSH_ON_DEVICE);

	//! \cond [create points] \endcond

	long int x_start = grid.size(0)*1.55f/domain.getHigh(0);
	long int y_start = grid.size(1)*1.55f/domain.getHigh(1);
	long int z_start = grid.size(1)*1.55f/domain.getHigh(2);

	long int x_stop = grid.size(0)*1.85f/domain.getHigh(0);
	long int y_stop = grid.size(1)*1.85f/domain.getHigh(1);
	long int z_stop = grid.size(1)*1.85f/domain.getHigh(2);

	//! \cond [create points sub] \endcond

	grid_key_dx<3> start({x_start,y_start,z_start});
	grid_key_dx<3> stop ({x_stop,y_stop,z_stop});

        grid.addPoints(start,stop,[] __device__ (int i, int j, int k)
                                {
                                                return true;
                                },
                                [] __device__ (InsertBlockT & data, int i, int j, int k)
                                {
                                        data.template get<U>() = 0.5;
                                        data.template get<V>() = 0.24;
                                }
                                );

	grid.template flush<smax_<U>,smax_<V>>(flush_type::FLUSH_ON_DEVICE);

	//! \cond [create points sub] \endcond
}


int main(int argc, char* argv[])
{
	openfpm_init(&argc,&argv);

	// domain
	Box<3,float> domain({0.0,0.0,0.0},{2.5,2.5,2.5});
	
	// grid size
        size_t sz[3] = {256,256,256};

	// Define periodicity of the grid
	periodicity<3> bc = {PERIODIC,PERIODIC,PERIODIC};
	
	// Ghost in grid unit
	Ghost<3,long int> g(1);
	
	// deltaT
	float deltaT = 0.25;

	// Diffusion constant for specie U
	float du = 2*1e-5;

	// Diffusion constant for specie V
	float dv = 1*1e-5;

	// Number of timesteps
#ifdef TEST_RUN
	size_t timeSteps = 300;
#else
        size_t timeSteps = 15000;
#endif

	// K and F (Physical constant in the equation)
    float K = 0.053;
    float F = 0.014;

	SparseGridType grid(sz,domain,g,bc);

	// spacing of the grid on x and y
	float spacing[3] = {grid.spacing(0),grid.spacing(1),grid.spacing(2)};

	init(grid,domain);

	// sync the ghost
	grid.template ghost_get<U,V>(RUN_ON_DEVICE);

	// because we assume that spacing[x] == spacing[y] we use formula 2
	// and we calculate the prefactor of Eq 2
	float uFactor = deltaT * du/(spacing[x]*spacing[x]);
	float vFactor = deltaT * dv/(spacing[x]*spacing[x]);

	auto & v_cl = create_vcluster();

	timer tot_sim;
	tot_sim.start();

	for (size_t i = 0; i < timeSteps ; ++i)
	{
		if (v_cl.rank() == 0)
		{std::cout << "STEP: " << i << std::endl;}
/*		if (i % 300 == 0)
		{
			std::cout << "STEP: " << i << std::endl;
			grid.write_frame("out",i,VTK_WRITER);
		}*/

		//! \cond [stencil get and use] \endcond

		typedef typename GetCpBlockType<decltype(grid),0,1>::type CpBlockType;

		//! \cond [lambda] \endcond

		auto func = [uFactor,vFactor,deltaT,F,K] __device__ (float & u_out, float & v_out,
				                                   CpBlockType & u, CpBlockType & v,
				                                   int i, int j, int k){

				float uc = u(i,j,k);
				float vc = v(i,j,k);

				u_out = uc + uFactor *(u(i-1,j,k) + u(i+1,j,k) +
                                                       u(i,j-1,k) + u(i,j+1,k) +
                                                       u(i,j,k-1) + u(i,j,k+1) - 6.0f*uc) - deltaT * uc*vc*vc
                                                       - deltaT * F * (uc - 1.0f);


				v_out = vc + vFactor *(v(i-1,j,k) + v(i+1,j,k) +
                                                       v(i,j+1,k) + v(i,j-1,k) +
                                                       v(i,j,k-1) + v(i,j,k+1) - 6.0f*vc) + deltaT * uc*vc*vc
					               - deltaT * (F+K) * vc;
				};

		//! \cond [lambda] \endcond

		//! \cond [body] \endcond

		if (i % 2 == 0)
		{
			hipDeviceSynchronize();
			timer tconv;
			tconv.start();
			grid.conv2<U,V,U_next,V_next,1>({0,0,0},{(long int)sz[0]-1,(long int)sz[1]-1,(long int)sz[2]-1},func);
			hipDeviceSynchronize();
			tconv.stop();
			std::cout << "Conv " << tconv.getwct() << std::endl;

			// After copy we synchronize again the ghost part U and V

			grid.ghost_get<U_next,V_next>(RUN_ON_DEVICE | SKIP_LABELLING);
		}
		else
		{
			grid.conv2<U_next,V_next,U,V,1>({0,0,0},{(long int)sz[0]-1,(long int)sz[1]-1,(long int)sz[2]-1},func);

			// After copy we synchronize again the ghost part U and V
			grid.ghost_get<U,V>(RUN_ON_DEVICE | SKIP_LABELLING);
		}

		//! \cond [body] \endcond

		// Every 500 time step we output the configuration for
		// visualization
//		if (i % 500 == 0)
//		{
//			grid.save("output_" + std::to_string(count));
//			count++;
//		}
	}
	
	tot_sim.stop();
	std::cout << "Total simulation: " << tot_sim.getwct() << std::endl;

	grid.deviceToHost<U,V,U_next,V_next>();
	grid.write("final");

	//! \cond [time stepping] \endcond

	/*!
	 * \page Grid_3_gs_3D_sparse Gray Scott in 3D
	 *
	 * ## Finalize ##
	 *
	 * Deinitialize the library
	 *
	 * \snippet Grid/3_gray_scott/main.cpp finalize
	 *
	 */

	//! \cond [finalize] \endcond

	openfpm_finalize();

	//! \cond [finalize] \endcond

	/*!
	 * \page Grid_3_gs_3D_sparse Gray Scott in 3D
	 *
	 * # Full code # {#code}
	 *
	 * \include Grid/3_gray_scott_3d/main.cpp
	 *
	 */
}

#else

int main(int argc, char* argv[])
{
        return 0;
}

#endif

